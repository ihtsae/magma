#include <stdio.h>
#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

//#define num_threads 64
#define dgemv_bs 32




__global__ void
l_ssymv_special (int n, float alpha ,  float* A, int lda, float *x, int ldx , float beta ,  float *y , int ldy ){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  float res = 0.f;


  __shared__ float buff [dgemv_bs];
  __shared__ float la   [dgemv_bs][dgemv_bs+1];


  A += ind;
  x += tx ;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    buff[tx]  = x[i];
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 

  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  x+= break_d ; 
  __syncthreads();
  //buff[tx]  = x[break_d];
  buff[tx]  = x[0];
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
	        la[tx][i] = la[i][tx] ; 
         }
	 else 
	        la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
  x-= break_d ; 
  break_d  += dgemv_bs ; 
  __syncthreads();




  for(int i=break_d; i<n; i += dgemv_bs ){
    buff[tx]  = x[i];
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
      __syncthreads();
  }


   la[tx][ty]= res ;
   __syncthreads();
   if( ty == 0 ) {
     res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
     y[ind] = beta * y[ind]  + alpha * res;
   }

}

__global__ void
l_ssymv_generic (int n, float alpha ,  float* A, int lda, float *x, int ldx , float beta ,  float *y , int ldy , int m_full_block , int m_mod_32){

  
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  float res = 0.f;


  __shared__ float buff [dgemv_bs];
  __shared__ float la   [dgemv_bs][dgemv_bs+1];

  if( blockIdx.x == m_full_block ) {
  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/
       if  ( tx < m_mod_32 ){
		A+= ( blockIdx.x * dgemv_bs + tx ) ;
       } 	 	
       else{
		A+= ( blockIdx.x * dgemv_bs + m_mod_32 -1) ; 
		//A+= ( blockIdx.x * dgemv_bs + 0) ; 
       }
       x+=tx;
       A+= ty * lda  ;  
       int break_d  =   blockIdx.x* dgemv_bs ;

	  /*----------------------------
		Go Right
	  -------------------------------*/

	  for(int  i=0; i<break_d; i += dgemv_bs ){
	    #pragma unroll 8 
	    for(int j=0; j < dgemv_bs ; j+=4){
	        la[tx][ty+j] = A[j*lda] ;
	    }
	    buff[tx]  = x[i];
	    __syncthreads();

	    #pragma unroll 8 
	    for(int j=0; j < 8 ; j++){
	       res+=la[tx][j+ty*8]*buff[j+ty*8];
	    }
	    A+=lda* dgemv_bs ;
	    __syncthreads(); 
	  }
	  /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
		x+= ( break_d -tx ) ; 
	        //buff[tx]  = x[i];
		/*--------------------------------------------
			he will compute the triangular parts
			others will be waiting with values. 
                -----------------------------------------------*/
		int j ;
                int count = 1 ; 
		if( tx < m_mod_32 ) 
			count = tx ; 
		else
			count = m_mod_32 ;
		for(j =0;j<=count;j++){
			res+= A[j*lda] * x[j];
                }
		A+=(tx)*lda;
		count = 1 ; 
		for(;j<m_mod_32;j++){
			res+= A[count] * x[j];
			count++;
		}
          }
          else{
          }
	  __syncthreads(); 
   	 la[tx][ty]= res ;
          __syncthreads();
         /*--------------------------------------------------------
	 The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
             res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
	     if( tx < m_mod_32)
                 y[ind] = beta * y[ind]  + alpha * res;
         }
	 
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  ****************************************
  -------------------------------------*/
  A += ind;
  x += tx ;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  /*----------------------------
	Go Right
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    buff[tx]  = x[i];
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
	Diagonal 
	Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  x+= break_d ; 
  __syncthreads();
  //buff[tx]  = x[break_d];
  buff[tx]  = x[0];
  /*--------------------------------------------
	Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
	        la[tx][i] = la[i][tx] ; 
         }
	 else 
	        la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
  /*--------------------------------
	Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
  x-= break_d ; 
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
	Go Down 
  -------------------------------*/
  for(int i=break_d; i<n; i += dgemv_bs ){
    buff[tx]  = x[i];
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
      __syncthreads();
  }

  
  /*---------------------------------------------
	doing m_mod_32 stuffs here.
	Symmetric is giving us benefit .. true
  -----------------------------------------------*/
    x-=tx;
    x+=n;
    A-=tx;
    if( tx < m_mod_32){
        buff[tx]  = x[tx];
	A+=tx;
    }
    else{
        buff[tx]  = 0*x[m_mod_32-1]; /*This will confirm valid memory reference*/
	A+=(m_mod_32-1); /* Same as above*/
    }

   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       if( tx < m_mod_32 ) 
       la[ty+j][tx] = 1.0 * A[ j * lda];
       else
       la[ty+j][tx] = 0.0 * A[ j * lda];
       
    }
    __syncthreads();

    /*----------------------------------------
	What about doing some Zeroing here?
	instead of zeroing before?
    -----------------------------------------*/	
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       //if( ( j+ty*8) < m_mod_32 )
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
    __syncthreads();
   

   la[tx][ty]= res ;
   __syncthreads();
   /*--------------------------------------------------------
	The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
     y[ind] = beta * y[ind]  + alpha * res;
   }

  }

}

__global__ void
u_ssymv_generic (int n, float alpha ,  float* A, int lda, float *x, int ldx , float beta ,  float *y , int ldy , int m_full_block , int m_mod_32){

  
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  float res = 0.f;


  __shared__ float buff [dgemv_bs];
  __shared__ float la   [dgemv_bs][dgemv_bs+1];
  int blockIdxx =  blockIdx.x ;

  if( blockIdx.x == m_full_block ) {

  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/

  ind =  tx ;
  A+= lda*(n-1) ; 
  x+= (n-1);


       if  ( tx < m_mod_32 ){
		A+= (  tx ) ;
       } 	 	
       else{
		A+= (  m_mod_32 -1) ; 
       }
       x-=tx;
       A-= ty * lda  ;  
       int break_d  =   (blockIdx.x)* dgemv_bs ;

	  /*----------------------------
		Go Right
	  -------------------------------*/

	  for(int  i=0; i<break_d; i += dgemv_bs ){
	    #pragma unroll 8 
	    for(int j=0; j < dgemv_bs ; j+=4){
	        la[tx][ty+j] = A[-j*lda] ;
	    }
	    buff[tx]  = x[-i];
	    __syncthreads();

	    #pragma unroll 8 
	    for(int j=0; j < 8 ; j++){
	       res+=la[tx][j+ty*8]*buff[j+ty*8];
	    }
	    A-=lda* dgemv_bs ;
	    __syncthreads(); 
	  }
	  /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
		x-= ( break_d -tx ) ; 
	        //buff[tx]  = x[i];
		/*--------------------------------------------
			he will compute the triangular parts
			others will be waiting with values. 
                -----------------------------------------------*/
		int j ;
                int count = 1 ; 
		if( tx < m_mod_32 ) 
			count =m_mod_32- tx ; 
		else
			count = m_mod_32 ;
		for(j =0;j<count;j++){
			res+= A[-j*lda] * x[-j];
                }
		A-=(count-1)*lda;
		count = 1 ; 
		for(;j<m_mod_32;j++){
			res+= A[-count] * x[-j];
			count++;
		}
          }
          else{
          }
	  __syncthreads(); 
   	 la[tx][ty]= res ;
          __syncthreads();
         /*--------------------------------------------------------
	 The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
             res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
	     if( tx < m_mod_32)
                 y[ind] = beta * y[ind]  + alpha * res;
         }
	 
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  -- By the way this code can be optimized more. 
  ****************************************
  -------------------------------------*/
  ind = blockIdx.x *  dgemv_bs + tx + m_mod_32 ;
  float *A1 = A ; 
  float *x1 = x ; 
  A+= lda*(n-1)  ; 
  x+= (n-1);

  A += ind;
  x -= tx ;
  A-= ty * lda  ;  

  int break_d  = (n / dgemv_bs -   blockIdxx-1 )* dgemv_bs ;
  /*----------------------------
	Go Left
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    buff[tx]  = x[-i];
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
	Diagonal 
	Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4){
         la[tx][31-ty-j] = A[ -j * lda];
  }

  A-= dgemv_bs ;
  x-= break_d ; 
  __syncthreads();
  buff[31-tx]  = x[0];
  /*--------------------------------------------
	Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
	        la[tx][i] = la[i][tx]; 
         }
	 else{ 
	        la[tx][i] = la[tx][i]  ;
	 }
  }
  __syncthreads();
  /*--------------------------------
	Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
  x+=break_d  ; 
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
	Go Up 
  -------------------------------*/
  int i ;
  for( i=break_d; i<n; i+= dgemv_bs ){
    buff[31-tx]  = x[-i] ;
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       la[ty+j][tx] = A[- j * lda];
    }
    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=la[31-tx][j+ty*8]*buff[j+ty*8];
    }
      __syncthreads();
  }
  /*---------------------------------------------
	doing m_mod_32 stuffs here.
	Symmetric is giving us benefit .. true
	Do the other way please......
  -----------------------------------------------*/
   A1 = A1 + m_mod_32 * lda + tx *lda ;  
   if( ty == 0  ) {
	for( int j = 0 ;  j < m_mod_32 ; j++){
		res+= x1[j] * A1[ j + lda * (blockIdx.x) * 32 ];
	}
   }
    __syncthreads();

   la[tx][ty]= res ;
   __syncthreads();
   /*--------------------------------------------------------
	The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
     y[ind] = beta * y[ind]  + alpha * res;
   }

  }

}






__global__ void
u_ssymv_special (int n, float alpha ,  float* A, int lda, float *x, int ldx , float beta ,  float *y , int ldy ){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  float res = 0.f;

  /*
	Reverse Computation ... 
		- Left 
		- Triangle 
		- Up 
  */

  A+= lda*(n-1) ; 
  x+= (n-1);
  __shared__ float buff [dgemv_bs];
  __shared__ float la   [dgemv_bs][dgemv_bs+1];


  A += ind;
  x -= tx ;
  A-= ty * lda  ;  
  int break_d  = (n / dgemv_bs -   blockIdx.x-1 )* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    buff[tx]  = x[-i];
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=la[tx][j+ty*8]*buff[j+ty*8];
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }




  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[tx][31-ty-j] = A[ -j * lda];
  /*
	Look at the indexing changes
  */

  A-= dgemv_bs ;
  x-= break_d ; 
  __syncthreads();
  buff[31-tx]  = x[0];
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
	        la[tx][i] = la[i][tx]; 
         }
	 else{ 
	        la[tx][i] = la[tx][i]  ;
	 }
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=la[tx][j+ty*8]*buff[j+ty*8];
    }

 x+=break_d  ; 
  break_d  += dgemv_bs ; 
  __syncthreads();



  for(int i=break_d; i<n; i+= dgemv_bs ){
    buff[31-tx]  = x[-i] ;
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ -j * lda];

    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=la[31-tx][j+ty*8]*buff[j+ty*8];
    }
      __syncthreads();
  }


   la[tx][ty]= res ;

   __syncthreads();
   if( ty == 0 ) {
     res = res + la[tx][1]+ la[tx][2]+ la[tx][3] ;
     y[ind] =  beta * y[ind]  + alpha * res;
   }

}





extern "C" void mssymv (char side , char uplo , int m , float alpha ,  float *A , int lda , 
 float *X , int incx , float beta , float *Y , int incy )
{
/*
Note:
	The UPLO = 'U' Version can be optimized more.
        side is not needed........................... 
*/
    int blocks;
    if (m % dgemv_bs==0)
        blocks = m/ dgemv_bs;
    else
        blocks = m/ dgemv_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 4, 1);

    if( m % dgemv_bs == 0 ) {
	    if( uplo == 'L' || uplo == 'l'){	
		    l_ssymv_special <<<grid, threads>>> (m, alpha , A, lda, X, incx ,beta,  Y , incy);
	    }
            else{
		    u_ssymv_special <<<grid, threads>>> (m, alpha , A, lda, X, incx ,beta,  Y , incy);
	    } 
		
    } 
    else{	
	    int  m_full_block = (m - m % 32 ) /32 ; 
	    int  m_mod_32 = m%32 ;  
	    if( uplo == 'L' || uplo == 'l'){
		    l_ssymv_generic <<<grid, threads>>> (m, alpha , A, lda, X, incx ,beta,  Y , incy, m_full_block , m_mod_32);
	    }	
	    else{
		    u_ssymv_generic <<<grid, threads>>> (m, alpha , A, lda, X, incx ,beta,  Y , incy, m_full_block , m_mod_32);
	    }	
    }
}


/*
Interface ..................................
*/

extern "C" void 
magma_ssymv (char side , char uplo , int m , float alpha ,  float *A , int lda ,  float *X , int incx , float beta , float *Y , int incy )
{
	mssymv (side, uplo , m , alpha , A , lda , X , incx , beta , Y , incy );
}
