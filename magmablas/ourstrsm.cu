/*
    -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009
*/

void CUDA_SAFE_CALL_US2( hipError_t err ) {
;
}


#include "hipblas.h"
//#include "cutil.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32 


__global__ void 
sgemvT32_kernel(int m, float alpha, float* A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A^T x where A is single precision 
    array of dimension (32, M).
*/

    const int inx = threadIdx.x;
    const int iny = threadIdx.y;

    int ind  = iny + __mul24(blockIdx.x,32);
    ind = inx + __mul24(ind,lda);
    int ind2 = inx + __mul24(iny,32);

    A += ind;
    x += inx;

    float res = 0.f;

    __shared__ float buff[64];
    __shared__ float la[32][33];

    buff[ind2]  = x[0];

    #pragma unroll
    for(int j=0; j<16; j++)
      la[iny+__mul24(2,j)][inx] = A[j*__mul24(2,lda)];

    __syncthreads();

    // multiply with the sub-matrix
    #pragma unroll
    for(int j=0; j <16; j++)
      res += la[inx][j+iny*16]*buff[j+iny*16];

    ind = inx + __mul24(blockIdx.x,32);
    la[inx][iny]= res;

    __syncthreads();

    if (ind<m){
       res = la[inx][0] + la[inx][1];
       y[ind] = alpha*res;
    }
}

__global__ void 
sgemv32_kernel(int n, float alpha, float* A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A x where A is single precision
    array of dimension (N, 32).
*/

    int ind = blockIdx.x*32 + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    float res = 0.f;

    __shared__ float buff[32];
    buff[threadIdx.x]  = x[0];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < 32; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }

    if (ind<n)
      y[ind] = alpha*res;
}

void magmablas_sgemv32(char tran, int n, float alpha, 
                       float *A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes 
       y = alpha A^T x           for tran = 'T' / 't' or
       y = alpha A x 
    where A is single precision array of dimension (32, N) for 
    tran = 'T' / 't', or of dimension (N, 32) otherwise.
*/

    int blocks;
    if (n % 32 == 0)
      blocks = n/32;
    else
      blocks = n/32 + 1;
    dim3 grid(blocks, 1, 1);

    if (tran == 'T' || tran == 't'){
      dim3 threads(32, 2, 1);
      sgemvT32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
    else 
    {
      dim3 threads(32, 1, 1);
      sgemv32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
}

__global__ void
inplace_sgemm_kernel_T(int M, float alpha, float *A, int lda, float *B, int ldb)
{
        int i;
        float myvalue1=0, myvalue2= 0 ;
        float med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,ty) + tx ;


        As[tx][ ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[tx][ty]= B[0];
        Bs[tx][ty+16]= B[16*ldb];

        __syncthreads();
        med  = As[tx][0];
        float py1 = Bs[ty][0] ;
        float py2 = Bs[ty+16][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                myvalue2 +=  med*py2;
                py2 = Bs[ty+16][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue1 +=  med*py1;
        myvalue2 +=  med*py2;
        A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
inplace_sgemm_kernel_N(int M, float alpha, float *A, int lda, float *B, int ldb)
{
        int i;
        float myvalue1=0, myvalue2= 0 ;
        float med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,ty) + tx ;


        As[tx][ ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[ty][tx]= B[0];
        Bs[ty+16][tx]= B[16*ldb];

        __syncthreads();
        med  = As[tx][0];
        float py1 = Bs[ty][0] ;
        float py2 = Bs[ty+16][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                myvalue2 +=  med*py2;
                py2 = Bs[ty+16][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue1 +=  med*py1;
        myvalue2 +=  med*py2;
        A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
diag_strtri_kernel (char uplo, char diag, float *A, float *d_dinvA, int lda)
{
	int i,j;
	float Ystx=0;
	float *Bw=NULL, *x=NULL, *y=NULL, *Aoff=NULL;
	float *my_d_dinvA;
	int switcher=0;

	// Thread index
	int tx = threadIdx.x;
	int txw;

	// Block index
	int bx = blockIdx.x;
		
	Aoff = A+bx*lda*BLOCK_SIZE+bx*BLOCK_SIZE;
	my_d_dinvA = d_dinvA+bx*BLOCK_SIZE*BLOCK_SIZE;

	__shared__ float Bs[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float workspace[BLOCK_SIZE]; // workspace used to store the current working column

	// load A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		Bs[i*BLOCK_SIZE+tx] = *(Aoff+i*lda+tx);	// read in the whole square block of my A
												// not the upper or lower diagonal

	// Synchronize to make sure the matrices are loaded
	__syncthreads();

	Bs[tx*BLOCK_SIZE+tx] = ((diag=='u' || diag=='U')?1:(1/Bs[tx*BLOCK_SIZE+tx]));	// solve the diagonals

	if (uplo == 'l' || uplo == 'L')
	{
		/*
		 * the lower case
		 */
		if (tx < BLOCK_SIZE-1)
			Bs[(BLOCK_SIZE-1)*BLOCK_SIZE+tx] = 0;	//zero out the last column, except the diagonal element

		for (i=BLOCK_SIZE-2; i>=0; i--)
		{
			Ystx = 0;
			switcher = (tx>i);
			
			//strmv
			Bw = Bs+(i+1)*BLOCK_SIZE+i+1;
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			x = workspace+i+1;
			y = Bs+i*BLOCK_SIZE;

			txw = (tx-i-1);

			#pragma unroll
			for (j=0; j<txw+1; j++)
				Ystx += (float)switcher*(*(Bw+j*BLOCK_SIZE+txw)*x[j]);

			//sscal
			switcher = (tx != i); 
			//if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = (float)switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+(float)(!switcher)*y[tx];

			__syncthreads();
		}

	}
	else
	{
		 /* the upper case */
		for (i=0; i<BLOCK_SIZE; i++)
		{
			Ystx = 0;
			switcher = (float)(tx<i);
			
			//strmv
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			y = Bs+i*BLOCK_SIZE;

			#pragma unroll
			for (j=tx; j<i; j++)
				Ystx += switcher*(*(Bs+j*BLOCK_SIZE+tx)*workspace[j]);

			//sscal
			switcher = (tx != i); // if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+!switcher*y[tx];

			__syncthreads();
		}


	}
		
	// write back A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		*(my_d_dinvA+i*BLOCK_SIZE+tx) = Bs[i*BLOCK_SIZE+tx];
}

extern "C" void
inplace_sgemm (char tran, int M, float alpha, float *A, int lda, float *B, int ldb)
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE/2);

	if (tran == 'n' || tran == 'N')
		inplace_sgemm_kernel_N<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
	else
		inplace_sgemm_kernel_T<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
}

/*
 * magmablas_strsmx
 * the expert interface
 */
void magmablas_strsmx ( char side, char uplo, char tran, char diag, int M, int N, float alpha, float* A, int lda, float* b, int ldb, float *d_dinvA)
{
/*  -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   STRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - REAL.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - REAL             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the strictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the strictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - REAL             array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.

		d_dinvA  REAL array of DIMENSION (BLOCKSIZE, M) when side='L', 
				 (BLOCKSIZE, N) when side='R'. On exit this space is filled
			     with the inverse of blocks on the diagonal, each inverse is
				 of size BLOCKSIZE x BLOCKSIZE, and the leading dimension of
				 d_dinvA is BLOCKSIZE;
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;

	if ((M%BLOCK_SIZE) != 0)
		return;

	if (d_dinvA == NULL)
		return;

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		nblocks = M/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				cublasSgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					cublasSgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
					return;

				cublasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
					return;

				cublasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				cublasSgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					cublasSgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		nblocks = N/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				cublasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				cublasSgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasSgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				cublasSgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasSgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				cublasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}
}

/*
 * magmablas_strsm
 */
extern "C"
void magmablas_strsm ( char side, char uplo, char tran, char diag, int M, int N, float* A, int lda, float* b, int ldb)
{
float alpha = 1.f ; 
/*  -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   STRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - REAL.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - REAL             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the strictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the strictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - REAL             array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;
	float *d_dinvA;

	if ((M%BLOCK_SIZE) != 0)
	{
		return;
	}

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		CUDA_SAFE_CALL_US2(hipMalloc((void**)&d_dinvA, BLOCK_SIZE*M*sizeof(float)));
		nblocks = M/BLOCK_SIZE;

		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					cublasSgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					cublasSgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		CUDA_SAFE_CALL_US2(hipMalloc((void**)&d_dinvA, BLOCK_SIZE*N*sizeof(float)));
		nblocks = N/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasSgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasSgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
					return;
				}

				cublasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					cublasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}
		
	CUDA_SAFE_CALL_US2(hipFree(d_dinvA));
}

