
#include <hip/hip_runtime.h>
/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009
*/

__global__ void ctranspose_32( float2 *B, int ldb, float2 *A, int lda )
{	
	__shared__ float2 a[32][33];
	
	int inx = threadIdx.x;
	int iny = threadIdx.y;
	int ibx = blockIdx.x*32;
	int iby = blockIdx.y*32;
	
	A += ibx + inx + __mul24( iby + iny, lda );
	B += iby + inx + __mul24( ibx + iny, ldb );
	
	a[iny+0][inx] = A[0*lda];
	a[iny+8][inx] = A[8*lda];
	a[iny+16][inx] = A[16*lda];
	a[iny+24][inx] = A[24*lda];
	
	__syncthreads();
	
	B[0*ldb] = a[inx][iny+0];
	B[8*ldb] = a[inx][iny+8];
	B[16*ldb] = a[inx][iny+16];
	B[24*ldb] = a[inx][iny+24];
} 

//
//	m, n - dimensions in the source matrix
//
extern "C" void 
magmablas_ctranspose(float2 *odata, int ldo, 
                    float2 *idata, int ldi, 
                    int m, int n )
{
	//assert( (m%32) == 0 && (n%32) == 0, "misaligned transpose" );
	dim3 threads( 32, 8, 1 );
	dim3 grid( m/32, n/32, 1 );
	ctranspose_32<<< grid, threads >>>( odata, ldo, idata, ldi );
}
