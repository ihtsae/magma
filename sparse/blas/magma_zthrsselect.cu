#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 32
#define GRID_SIZE1 32768
#define GRID_SIZE2 1024
#define GRID_SIZE3 32
#define GRID_SIZE4 1


// kernel for counting elements for different thresholds
__global__ void 
zthreshselect_kernel( 
    magma_int_t total_size,
    magma_int_t subset_size,
    magmaDoubleComplex *val,
    float *thrs )
{
    int tidx = threadIdx.x;   
    int bidx = blockIdx.x;
    int gtidx = bidx * blockDim.x + tidx;
    
    // now define the threshold
    float thrs_loc = ((float)(gtidx)) / ((float) ( blockDim.x*gridDim.x ) );
    float lval;
    
    // local counter
    magma_int_t count = 0;
    

    //printf("threshold[%d] = %.8e\n", gtidx, thrs_loc);
    for (magma_int_t z=0; z<total_size; z+=32) {
        lval = MAGMA_Z_ABS(val[z]);
        count = (lval < thrs_loc) ? count+1 : count;
        #if __CUDA_ARCH__ >= 300
        #if __CUDACC_VER_MAJOR__ < 9
            #pragma unroll
            for (int z=0; z<31; z++) {
                lval = __shfl_down(lval, 1);
                count = (lval < thrs_loc) ? count+1 : count;
            }
        #else
            #pragma unroll
            for (int z=0; z<31; z++) {
                lval = __shfl_down_sync(0xffffffff,lval, 1, 32);
                count = (lval < thrs_loc) ? count+1 : count;
            }
        #endif
        #endif
    }
    
    // if count > subset_size -> thrs_loc = 0.0
    thrs_loc = (count < subset_size) ?  thrs_loc : 0.0;
    
    // check for the largest threshold in warp and write out
    float maxval = thrs_loc;
    
    #if __CUDA_ARCH__ >= 300
    #if __CUDACC_VER_MAJOR__ < 9
        #pragma unroll
        for (int z=0; z<31; z++) {
            thrs_loc = __shfl_down(thrs_loc, 1);
            maxval = thrs_loc > maxval ? thrs_loc : maxval ;
        }
    #else
        #pragma unroll
        for (int z=0; z<31; z++) {
            thrs_loc = __shfl_down_sync(0xffffffff,thrs_loc, 1, 32);
            maxval = thrs_loc > maxval ? thrs_loc : maxval ;
        }
    #endif
    #endif
    
    if ( tidx == 0 ){
            thrs[bidx] = maxval;
    }
}


// kernel identifying the best threshold
__global__ void
magma_zreduce_thrs( 
    float *thrs,
    float *thrs2)
{
    int tidx = threadIdx.x;   
    int bidx = blockIdx.x;
    int gtidx = bidx * blockDim.x + tidx;
    
    float val = thrs[gtidx];
    float maxval = val;
    
#if __CUDA_ARCH__ >= 300
#if __CUDACC_VER_MAJOR__ < 9
    #pragma unroll
    for (int z=0; z<31; z++) {
        val = __shfl_down(val, 1);
        maxval = val > maxval ? val : maxval ;
    }
#else
    #pragma unroll
    for (int z=0; z<31; z++) {
        val = __shfl_down_sync(0xffffffff,val, 1, 32);
        maxval = val > maxval ? val : maxval ;
    }
#endif
#endif

    if ( tidx == 0 ){
            thrs2[bidx] = maxval;
    }
}



/**
    Purpose
    -------
    
    This routine selects a threshold separating the subset_size smallest
    magnitude elements from the rest.
    Hilarious approach: 
    Start a number of threads, each thread uses a pre-defined threshold, then
    checks for each element whether it is smaller than the threshold.
    In the end a global reduction identifies the threshold that is closest.

    Assuming all values are in (0,1), the distinct thresholds are defined as:
    
    threshold [ thread ] = thread / num_threads
    
    We obviously need to launch many threads.
    
    Arguments
    ---------
                
    @param[in]
    total_size  magma_int_t
                size of array val
                
    @param[in]
    subset_size magma_int_t
                number of smallest elements to separate
                
    @param[in]
    val         magmaDoubleComplex
                array containing the values
                
    @param[out]
    thrs        float*  
                computed threshold

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zthrsholdselect(
    magma_int_t total_size,
    magma_int_t subset_size,
    magmaDoubleComplex *val,
    double *thrs,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    
    dim3 block(BLOCK_SIZE, 1, 1);
    
    dim3 grid1(GRID_SIZE1, 1, 1 );
    dim3 grid2(GRID_SIZE2, 1, 1 );
    dim3 grid3(GRID_SIZE3, 1, 1 );
    dim3 grid4(GRID_SIZE4, 1, 1 );
    
    float *thrs1, *thrs2, *thrstmp; 
    
    CHECK(magma_smalloc_cpu(&thrstmp, 1));
    CHECK(magma_smalloc(&thrs1, GRID_SIZE1));
    CHECK(magma_smalloc(&thrs2, GRID_SIZE2));
    
    // first kernel checks how many elements are smaller than the threshold
    zthreshselect_kernel<<<grid1, block, 0, queue->hip_stream()>>>
        (total_size, subset_size, val, thrs1);
        
    // second kernel identifies the largest of these thresholds
    magma_zreduce_thrs<<<grid2, block, 0, queue->hip_stream()>>>
        ( thrs1, thrs2 );
    magma_zreduce_thrs<<<grid3, block, 0, queue->hip_stream()>>>
        ( thrs2, thrs1 );
    magma_zreduce_thrs<<<grid4, block, 0, queue->hip_stream()>>>
        ( thrs1, thrs2 );
        
    magma_sgetvector(1, thrs2, 1, thrstmp, 1, queue );
    
    thrs[0] = (double)thrstmp[0];
    
cleanup:
    magma_free(thrs1);
    magma_free(thrs2);
    magma_free_cpu(thrstmp);

    return info;
}
