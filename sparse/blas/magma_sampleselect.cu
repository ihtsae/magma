#include "hip/hip_runtime.h"
#include "magma_sampleselect.h"

namespace magma_sampleselect {

__global__ void reduce_counts(const magma_int_t* __restrict__ in,
                              magma_int_t* __restrict__ out,
                              magma_int_t num_blocks) {
    magma_int_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < searchtree_width) {
        magma_int_t sum{};
        for (magma_int_t i = 0; i < num_blocks; ++i) {
            sum += in[idx + i * searchtree_width];
        }
        out[idx] = sum;
    }
}

__global__ void prefix_sum_counts(magma_int_t* __restrict__ in,
                                  magma_int_t* __restrict__ out,
                                  magma_int_t num_blocks) {
    magma_int_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < searchtree_width) {
        magma_int_t sum{};
        for (magma_int_t i = 0; i < num_blocks; ++i) {
            auto tmp = in[idx + i * searchtree_width];
            in[idx + i * searchtree_width] = sum;
            sum += tmp;
        }
        out[idx] = sum;
    }
}

__global__ void sampleselect_findbucket(magma_int_t* __restrict__ totalcounts,
                                        magma_int_t rank,
                                        unsigned* __restrict__ out_bucket,
                                        magma_int_t* __restrict__ out_rank) {
    prefix_sum_select(totalcounts, rank, out_bucket, out_rank);
}

magma_int_t sampleselect_alloc_size(magma_int_t size) {
    static_assert(sizeof(magma_int_t) >= sizeof(unsigned), "c++ is broken");
    return 1 // bucket index
         + 1 // rank
         + 1 // atomic
         + searchtree_width   // totalcount
         + num_grouped_blocks
           * searchtree_width // localcount
         + (size + 3) / 4;    // oracles
}

magma_int_t realloc_if_necessary(magma_ptr *ptr, magma_int_t *size, magma_int_t required_size) {
    magma_int_t info = 0;
    if (*size < required_size) {
        auto newsize = required_size * 5 / 4;
        CHECK(magma_free(*ptr));
        CHECK(magma_malloc(ptr, newsize));
        *size = newsize;
    }

cleanup:
    return info;
}

} // namespace magma_sampleselect